#include "hip/hip_runtime.h"
#include "Cuda.cuh"

#define THREADS 1024
#define BLOCKS(X) (X + THREADS - 1) / THREADS
#define UNREACHED -1


__global__ int FordFulkersonCuda::ComputeNextQueue(int to_pop_num,  int * pop_queue, int * push_queue, int * flow_matrix, bool * visited, int *parent_node) {
    // push_queue attualmente ci pensa la cpu
    // senò il primo thread che entra
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < to_pop_num) {
        const int current = pop_queue[tid];
        for (int adj = 0;  adj < d_nodes_num; adj++) {
            const int adj_matrix_index = current * d_nodes_num + adj;

            if (flow_matrix[adj_matrix_index] > 0 && !visited[adj]) {
                const int push_at = atomicAdd(&pushed_num, 1) - 1;
                push_queue[push_at] = adj;
                atomicExch(parent_node + adj, current);
                visited[adj] = true;
            }
        }
    }

    return pushed_num;
}


void FordFulkersonCuda::InitializeGraphOnDevice(Graph *g)
{
    const int nodes_num = g->GetNodesNumber();
    unsigned int bytes_matrix = nodes_num * nodes_num * sizeof(nodes_num);
    vector<Node> nodes = g->GetNodes();

    hipMalloc((void **)this->d_flow_matrix, bytes_matrix);
    hipMemset((void **)this->d_flow_matrix, 0, bytes_matrix);
    hipDeviceSynchronize();

    for (int start = 0; start < nodes_num; start++) {
        
        for (DirectedEdge<Node> e : *(nodes[start].GetEdges())) {
            const int end = e.GetEnd()->GetNodeNum();
            const int max_flow = e.GetMaxFlow();
            const int edge_index = start * nodes_num + end;
            int *d_edge_index = this->d_flow_matrix + edge_index;

            hipMemset((void **)d_edge_index, max_flow, sizeof(max_flow));
        }
    }
}

void FordFulkersonCuda::InitializeParentNode() {
    hipMalloc((void **)this->d_parent_node, this->nodes_num * sizeof(int));
    hipMemset((void **)this->d_parent_node, UNREACHED, this->nodes_num * sizeof(int));
}


void FordFulkersonCuda::InitializeVisited() {
    hipMalloc((void **)this->d_visited, this->nodes_num * sizeof(bool));
    hipMemset((void **)this->d_visited, false, this->nodes_num * sizeof(bool));
}


int FordFulkersonCuda::FindMinFlow() {
    /* computes min flow in path found */
    int min_flow = 0;

    return min_flow;
}


PathCuda FordFulkersonCuda::BFS(Node *start, Node *end) {
    int *d_pop_queue = this->d_first_queue;
    int *d_push_queue = this->d_second_queue;
    const int end_num = end->GetNodeNum();

    hipMemset((void **)this->d_parent_node + end_num, UNREACHED, sizeof(int));
    
    const int start_num = start->GetNodeNum();
    hipMemset((void **)d_pop_queue, start_num, sizeof(start_num));

    this->pushed_num = 1;

    hipDeviceSynchronize();

    while (this->pushed_num > 0) {
        const int pop_num = this->pushed_num;
        this->pushed_num = 0;

        ComputeNextQueue<<<BLOCKS(pop_num), THREADS>>>(pop_num, d_pop_queue, d_push_queue, d_flow_matrix, d_visited, d_parent_node);
        
        hipDeviceSynchronize();

        int * tmp = d_pop_queue;
        d_pop_queue = d_push_queue;
        d_push_queue = tmp;
    }

    return PathCuda(end_num);
}


FordFulkersonCuda::FordFulkersonCuda(Graph * g) : FordFulkersonSerial(g) {
    this->nodes_num = g->GetNodesNumber();
    this->InitializeGraphOnDevice(g);
    this->InitializeParentNode();
    this->InitializeVisited();
    hipDeviceSynchronize();

    this->d_nodes_num = this->nodes_num;
}


int FordFulkersonCuda::Solve() {
    Node * source = this->copy->GetSource();
    Node * sink = this->copy->GetSilk();
    int max_flow = 0;
    
    PathCuda bfs_path = this->BFS(source, sink);

    while (!bfs_path.IsEmpty()) {

        max_flow += bfs_path.IncreaseFlow();
        bfs_path = this->BFS(source, sink);
    
    }

    return max_flow;
}


FordFulkersonCuda::~FordFulkersonCuda() {
}

__global__ int PathCuda::IncreaseFlowKernel(int path_len) {
}

__global__ int PathCuda::FindPath(int end) {
    int len = 0;

}

PathCuda::PathCuda(int end) {
    this->path_length = 0;
    int current_node = end;
    hipMalloc((void **)this->d_min_flow, sizeof(int));
    hipMemset(this->d_min_flow, __INT_MAX__, sizeof(int));

    this->path_length = FindPath<<<1,1>>>(end);
    
}

bool PathCuda::IsEmpty() {
    return this->path_length == 0;
}

int PathCuda::IncreaseFlow() {
    return IncreaseFlowKernel<<<BLOCKS(path_length), THREADS>>>(this->path_length);
}

PathCuda::~PathCuda() {
    hipFree(this->d_min_flow);
}