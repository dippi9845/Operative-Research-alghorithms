#include "hip/hip_runtime.h"
#include "Cuda.cuh"
#include <cstdio>

#define CUDA_SAFE_CALL(call)                                                                       \
    do {                                                                                           \
        hipError_t cudaStatus = (call);                                                           \
        if (cudaStatus != hipSuccess) {                                                            \
            printf("Errore CUDA in %s:%d -- %s\n", __FILE__, __LINE__, hipGetErrorString(cudaStatus)); \
            exit(EXIT_FAILURE);                                                                    \
        }                                                                                          \
    } while (0)

#define THREADS 1024
#define BLOCKS(X) (X + THREADS - 1) / THREADS
#define UNREACHED -1

__managed__ int d_nodes_num;
__managed__ int mng_pushed_num;

__managed__ int mng_flow;
__managed__ int mng_path_len;

__global__ void IncreaseFlowKernel(int * start_node, int * destination_node, int * flow_matrix) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if ( tid < mng_path_len ) {
        const int start = start_node[tid];
        const int dest = destination_node[tid];
        const int flow_idx = start * d_nodes_num + dest;
        flow_matrix[flow_idx] -= mng_flow;
    }
}

__global__ void FindPath(int end, int * parent_node, int * start_node, int * destination_node, int * flow_matrix) {
    int current = end;
    mng_path_len = 0;
    mng_flow = __INT_MAX__;

    while (parent_node[current] != UNREACHED) {
        const int start = parent_node[current];
        const int destination = current;

        start_node[mng_path_len] = start;
        destination_node[mng_path_len] = destination;

        const int flow_idx = start * d_nodes_num + destination;

        if (mng_flow > flow_matrix[flow_idx]) {
            mng_flow = flow_matrix[flow_idx];
        }

        mng_path_len++;

        current = parent_node[current];
    }
}

__global__ void ComputeNextQueue(int to_pop_num,  int * pop_queue, int * push_queue, int * flow_matrix, bool * visited, int *parent_node) {
    // push_queue attualmente ci pensa la cpu
    // senò il primo thread che entra
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < to_pop_num) {
        const int current = pop_queue[tid];
        for (int adj = 0;  adj < d_nodes_num; adj++) {
            const int adj_matrix_index = current * d_nodes_num + adj;

            if (flow_matrix[adj_matrix_index] > 0 && !visited[adj]) {
                visited[adj] = true;
                const int push_at = atomicAdd(&mng_pushed_num, 1) - 1;
                push_queue[push_at] = adj;
                atomicExch(parent_node + adj, current);
            }
        }
    }
}



void FordFulkersonCuda::InitializeGraphOnDevice(Graph *g)
{
    const int nodes_num = g->GetNodesNumber();
    unsigned int bytes_matrix = nodes_num * nodes_num * sizeof(nodes_num);
    vector<Node> nodes = g->GetNodes();

    CUDA_SAFE_CALL(hipMalloc((void **)d_flow_matrix, bytes_matrix));
    hipMemset((void **)d_flow_matrix, 0, bytes_matrix);
    hipDeviceSynchronize();

    for (int start = 0; start < nodes_num; start++) {
        
        for (DirectedEdge<Node> e : *(nodes[start].GetEdges())) {
            const int end = e.GetEnd()->GetNodeNum();
            const int max_flow = e.GetMaxFlow();
            const int edge_index = start * nodes_num + end;
            int *d_edge_index = this->d_flow_matrix + edge_index;

            hipMemset((void **)d_edge_index, max_flow, sizeof(max_flow));
        }
    }
}

void FordFulkersonCuda::InitializeStartDestination() {
    hipMalloc((void **)this->d_start_node, this->nodes_num * sizeof(int));
    hipMalloc((void **)this->d_destination_node, this->nodes_num * sizeof(int));
}

void FordFulkersonCuda::InitializeParentNode() {
    hipMalloc((void **)this->d_parent_node, this->nodes_num * sizeof(int));
    hipMemset((void **)this->d_parent_node, UNREACHED, this->nodes_num * sizeof(int));
}


void FordFulkersonCuda::InitializeVisited() {
    hipMalloc((void **)this->d_visited, this->nodes_num * sizeof(bool));
    hipMemset((void **)this->d_visited, false, this->nodes_num * sizeof(bool));
}


bool FordFulkersonCuda::BFS(Node *start, Node *end) {
    int *d_pop_queue = this->d_first_queue;
    int *d_push_queue = this->d_second_queue;
    const int end_num = end->GetNodeNum();

    hipMemset((void **)this->d_parent_node + end_num, UNREACHED, sizeof(int));
    
    const int start_num = start->GetNodeNum();
    hipMemset((void **)d_pop_queue, start_num, sizeof(start_num));

    // settare lo start come visited
    // azzerare visited

    mng_pushed_num = 1;

    hipDeviceSynchronize();

    while (mng_pushed_num > 0) {
        const int pop_num = mng_pushed_num;
        mng_pushed_num = 0;

        ComputeNextQueue<<<BLOCKS(pop_num), THREADS>>>(pop_num, d_pop_queue, d_push_queue, d_flow_matrix, d_visited, d_parent_node);
        
        hipDeviceSynchronize();

        int * tmp = d_pop_queue;
        d_pop_queue = d_push_queue;
        d_push_queue = tmp;
    }

    FindPath<<<1,1>>>(end_num, d_parent_node, d_start_node, d_destination_node, d_flow_matrix);

    return mng_path_len > 0;
}


FordFulkersonCuda::FordFulkersonCuda(Graph * g) {
    this->graph = g;
    this->nodes_num = g->GetNodesNumber();
    this->InitializeGraphOnDevice(g);
    this->InitializeParentNode();
    this->InitializeVisited();
    this->InitializeStartDestination();
    hipDeviceSynchronize();

    d_nodes_num = this->nodes_num;
}


int FordFulkersonCuda::Solve() {
    Node * source = this->graph->GetSource();
    Node * sink = this->graph->GetSilk();
    int max_flow = 0;

    while (this->BFS(source, sink)) {
        IncreaseFlowKernel<<<BLOCKS(mng_path_len), THREADS>>>(d_start_node, d_destination_node, d_flow_matrix);
        max_flow += mng_flow;
    }

    return max_flow;
}


FordFulkersonCuda::~FordFulkersonCuda() {
}