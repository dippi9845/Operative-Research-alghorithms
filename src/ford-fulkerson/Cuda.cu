#include "hip/hip_runtime.h"
#include "Cuda.cuh"
#include <cstdio>

#define CUDA_SAFE_CALL(call)                                                                       \
    do {                                                                                           \
        hipError_t cudaStatus = (call);                                                           \
        if (cudaStatus != hipSuccess) {                                                            \
            printf("CUDA Error at %s:%d -- %s\n", __FILE__, __LINE__, hipGetErrorString(cudaStatus)); \
            exit(EXIT_FAILURE);                                                                    \
        }                                                                                          \
    } while (0)

#define THREADS 1024
#define BLOCKS(X) (X + THREADS - 1) / THREADS
#define UNREACHED -1

__managed__ int d_nodes_num;
__managed__ int mng_pushed_num;

__managed__ int mng_flow;
__managed__ int mng_path_len;

__global__ void IncreaseFlowKernel(int * start_node, int * destination_node, int * flow_matrix) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if ( tid < mng_path_len ) {
        const int start = start_node[tid];
        const int dest = destination_node[tid];
        const int flow_idx = start * d_nodes_num + dest;
        flow_matrix[flow_idx] -= mng_flow;
    }
}

__global__ void FindPath(int end, int * parent_node, int * start_node, int * destination_node, int * flow_matrix) {
    int current = end;
    mng_path_len = 0;
    mng_flow = __INT_MAX__;

    while (parent_node[current] != UNREACHED) {
        const int start = parent_node[current];
        const int destination = current;
        start_node[mng_path_len] = start;
        destination_node[mng_path_len] = destination;

        const int flow_idx = start * d_nodes_num + destination;

        if (mng_flow > flow_matrix[flow_idx]) {
            mng_flow = flow_matrix[flow_idx];
        }

        mng_path_len++;

        current = parent_node[current];
    }
}

__global__ void ComputeNextQueue(int to_pop_num,  int * pop_queue, int * push_queue, int * flow_matrix, unsigned int * visited, int *parent_node) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < to_pop_num) {
        const int current = pop_queue[tid];

        for (int adj = 0;  adj < d_nodes_num; adj++) {
            const int adj_matrix_index = current * d_nodes_num + adj;
            if (flow_matrix[adj_matrix_index] > 0 && visited[adj] == 0) {
                if (atomicAdd(visited + adj, 1) == 0) {
                    visited[adj] = true;
                    const int push_at = atomicAdd(&mng_pushed_num, 1);
                    push_queue[push_at] = adj;
                    atomicExch(parent_node + adj, current);

                }
            }
        }
    }
}



void FordFulkersonCuda::InitializeGraphOnDevice(Graph *g)
{
    const int nodes_num = g->GetNodesNumber();
    unsigned int bytes_matrix = nodes_num * nodes_num * sizeof(nodes_num);
    vector<Node> nodes = g->GetNodes();

    CUDA_SAFE_CALL(hipMalloc((void **)&d_flow_matrix, bytes_matrix));
    CUDA_SAFE_CALL(hipMemset(d_flow_matrix, 0, bytes_matrix));
    hipDeviceSynchronize();

    for (int start = 0; start < nodes_num; start++) {
        for (DirectedEdge<Node> e : *(nodes[start].GetEdges())) {
            const int end = e.GetEnd()->GetNodeNum();
            const int max_flow = e.GetMaxFlow();
            const int edge_index = start * nodes_num + end;
            int *d_edge_index = this->d_flow_matrix + edge_index;

            CUDA_SAFE_CALL(hipMemcpy(d_edge_index, &max_flow, sizeof(max_flow), hipMemcpyHostToDevice));
        }
    }
}

void FordFulkersonCuda::InitializeQueues() {
    CUDA_SAFE_CALL(hipMalloc((void **)&this->d_first_queue, this->nodes_num * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void **)&this->d_second_queue, this->nodes_num * sizeof(int)));
}

void FordFulkersonCuda::InitializeStartDestination() {
    CUDA_SAFE_CALL(hipMalloc((void **)&this->d_start_node, this->nodes_num * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void **)&this->d_destination_node, this->nodes_num * sizeof(int)));
}

void FordFulkersonCuda::InitializeParentNode() {
    CUDA_SAFE_CALL(hipMalloc((void **)&this->d_parent_node, this->nodes_num * sizeof(int)));
    CUDA_SAFE_CALL(hipMemset(this->d_parent_node, UNREACHED, this->nodes_num * sizeof(int)));
}


void FordFulkersonCuda::InitializeVisited() {
    CUDA_SAFE_CALL(hipMalloc((void **)&this->d_visited, this->nodes_num * sizeof(unsigned int)));
    CUDA_SAFE_CALL(hipMemset(this->d_visited, 0, this->nodes_num * sizeof(bool)));
}


bool FordFulkersonCuda::BFS(Node *start, Node *end) {
    int *d_pop_queue = this->d_first_queue;
    int *d_push_queue = this->d_second_queue;
    const int end_num = end->GetNodeNum();

    CUDA_SAFE_CALL(hipMemset(this->d_parent_node + end_num, UNREACHED, sizeof(int)));

    int start_num = start->GetNodeNum();
    CUDA_SAFE_CALL(hipMemcpy(d_pop_queue, &start_num, sizeof(int), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemset(d_visited, 0, sizeof(unsigned int) * this->nodes_num));
    CUDA_SAFE_CALL(hipMemset(d_visited + start_num, 1, sizeof(bool)));


    mng_pushed_num = 1;

    CUDA_SAFE_CALL(hipDeviceSynchronize());

    while (mng_pushed_num > 0) {
        const int pop_num = mng_pushed_num;
        mng_pushed_num = 0;

        ComputeNextQueue<<<BLOCKS(pop_num), THREADS>>>(pop_num, d_pop_queue, d_push_queue, d_flow_matrix, d_visited, d_parent_node);

        CUDA_SAFE_CALL(hipDeviceSynchronize());

        int * tmp = d_pop_queue;
        d_pop_queue = d_push_queue;
        d_push_queue = tmp;
    }

    FindPath<<<1,1>>>(end_num, d_parent_node, d_start_node, d_destination_node, d_flow_matrix);
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    return mng_path_len > 0;
}


FordFulkersonCuda::FordFulkersonCuda(Graph * g) {
    this->graph = g;
    this->nodes_num = g->GetNodesNumber();
    this->InitializeGraphOnDevice(g);
    this->InitializeQueues();
    this->InitializeParentNode();
    this->InitializeVisited();
    this->InitializeStartDestination();
    hipDeviceSynchronize();

    d_nodes_num = this->nodes_num;
}


int FordFulkersonCuda::Solve() {
    Node * source = this->graph->GetSource();
    Node * sink = this->graph->GetSilk();
    int max_flow = 0;

    while (this->BFS(source, sink)) {
        IncreaseFlowKernel<<<BLOCKS(mng_path_len), THREADS>>>(d_start_node, d_destination_node, d_flow_matrix);
        max_flow += mng_flow;
    }

    return max_flow;
}


FordFulkersonCuda::~FordFulkersonCuda() {
    CUDA_SAFE_CALL(hipFree(d_flow_matrix));
    CUDA_SAFE_CALL(hipFree(d_parent_node));
    CUDA_SAFE_CALL(hipFree(d_visited));
    CUDA_SAFE_CALL(hipFree(d_first_queue));
    CUDA_SAFE_CALL(hipFree(d_second_queue));
    CUDA_SAFE_CALL(hipFree(d_destination_node));
    CUDA_SAFE_CALL(hipFree(d_start_node));
    CUDA_SAFE_CALL(hipDeviceSynchronize());
}